
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// current time: will return the current time.    
double currentTime() {
  struct timeval now;
  gettimeofday(&now, NULL);

  return now.tv_sec + now.tv_usec/1000000.0;
}
