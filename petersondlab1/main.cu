#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "timing.h"

typedef unsigned long long bignum;

__host__ int isPrime(bignum num);
__device__ int disPrime(bignum num);
void checkPrimes(char * result, bignum num, bignum n);
int totalPrimes(char * arr, int size);
__global__ void dCheckPrimes(char * result);

int main() {
  double now, then;
  double scost, pcost;
  bignum blockSize;
  blockSize = 1024;


  // bignum test = 1000.00;
  bignum *h_n, *h_s =  (bignum *) malloc(sizeof(bignum));
  h_n = (bignum *) malloc(sizeof(bignum));
  *h_n = 1000000.00;
  *h_s = 0.00;
  char * result = (char *) malloc((*h_n + 1) *sizeof(int));
  char * d_result = (char *) malloc((*h_n + 1) *sizeof(int));

  then = currentTime();
  checkPrimes(result, 0, *h_n);
  now = currentTime();
  scost = (now - then) * 1000;
  printf("time taken calculating primes using cpu: %lf ms\n", scost);
  int tempPrime = totalPrimes(result, (*h_n + 1));
  printf("Total primes found: %d\n", tempPrime);

  then = currentTime();
  hipMalloc((void**) &d_result, *h_n * sizeof(int));
  hipMemcpy( d_result, result, *h_n * sizeof(int), hipMemcpyHostToDevice);
  dCheckPrimes<<<ceil((*h_n+1)/2.0/blockSize),blockSize>>>( d_result);
  hipMemcpy( result, d_result, *h_n * sizeof(int), hipMemcpyDeviceToHost);
  now = currentTime();
  pcost = (now - then) * 1000;
  printf("time taken calculating primes using cuda: %lf ms\n", pcost);
  tempPrime = totalPrimes(result, (*h_n + 1));
  printf("Total primes found: %d\n", tempPrime);

  hipFree(d_result);
  free(result);
}

/*
  dCheckPrimes: kernal function used to start operations on the gpu using cuda

  result -> the array that the function will return the results too
*/
__global__ void dCheckPrimes(char * result) {
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  if (id == 0) { id += 2;}
  else { id = id + id + 1;}
  result[id] = disPrime(id);
}

/*
  checkPrimes: function used to find primes in an array using the cpu

  result -> the array that the function will return the results too
  n -> number to check for prime up too
*/
void checkPrimes(char * result, bignum num, bignum n) {
  bignum i;

  if(num % 2 == 0) { num++; }

  for(i=num; i<num+n; i = i+2) {
    result[i] = isPrime(i);
  }

}

/*
  isPrime: takes one integer and checks if integer is prime or not used by host

  num -> integer to check if prime
*/
__host__ int isPrime(bignum num) {
  
  bignum i;
  bignum lim = (bignum) sqrt(num) + 1;

  for(i = 2; i<lim; i++) {
    if(num % i == 0) {
      return 0;
    }
  }
  return 1;
}

/*
  disPrime: takes one integer and checks if integer is prime or not used by device

  num -> integer to check if prime
*/
__device__ int disPrime(bignum num) {

  bignum i;
  bignum lim = (bignum) sqrtf(num) + 1;

  for(i = 2; i<lim; i++) {
    if(num % i == 0) {
      return 0;
    }
  }
  return 1;
}

/*
  totalPrimes: function used to check for total amount of primes in array

  arr -> array filled with primes to count
  size -> size of array being passed in 
*/
int totalPrimes(char * arr, int size) {
  int j = 0;
  for(int i = 0; i < size; i++) {
    if(arr[i] == 1) {
      j++;
    }
  }

  return j;
}